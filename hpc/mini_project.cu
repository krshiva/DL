#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void knapsack(int n, int capacity, int* weights, int* values, int* solution, int* start) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= capacity + 1) {
        return;
    }
    for (int i = start[tid]; i < n; i++) {
        if (weights[i] <= tid) {
            int temp = solution[tid - weights[i]] + values[i];
            if (temp > solution[tid]) {
                solution[tid] = temp;
                start[tid] = i;
            }
        }
    }
}

int main() {
    int n = 5;
    int capacity = 10;
    int weights[] = {2, 3, 5, 4, 1};
    int values[] = {5, 6, 10, 9, 2};

    int* gpu_capacity, *gpu_weights, *gpu_values, *gpu_solution, *gpu_start;
    hipMalloc(&gpu_capacity, sizeof(int));
    hipMalloc(&gpu_weights, n * sizeof(int));
    hipMalloc(&gpu_values, n * sizeof(int));
    hipMalloc(&gpu_solution, (capacity + 1) * sizeof(int));
    hipMalloc(&gpu_start, (capacity + 1) * sizeof(int));

    hipMemcpy(gpu_capacity, &capacity, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_weights, weights, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_values, values, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (capacity + threadsPerBlock - 1) / threadsPerBlock;
    knapsack <<<blocksPerGrid, threadsPerBlock>>>(n, capacity, gpu_weights, gpu_values, gpu_solution, gpu_start);

    int* solution = new int[capacity + 1];
    hipMemcpy(solution, gpu_solution, (capacity + 1) * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Maximum Value: " << solution[capacity] << endl;

    hipFree(gpu_capacity);
    hipFree(gpu_weights);
    hipFree(gpu_values);
    hipFree(gpu_solution);
    hipFree(gpu_start);
    delete[] solution;
    return 0;
}
